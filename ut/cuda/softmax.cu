#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <stdio.h>

__global__ void softmax1st(float* input,  float* mul,
                               float* param,  float* output) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  // blockDim.x == 256
  int output_dim = blockIdx.x;
  int thread_id = threadIdx.x;
  int warp_id = thread_id / 32;
  int lane_id = thread_id % 32;

  float local_max = -INFINITY;
  for (int64_t i = thread_id; i < 1024; i += blockDim.x) {
      float tmp = input[output_dim * 1024 + i] * param[i];
      mul[output_dim * 1024 + i] = tmp;
      if (tmp > local_max) {
          local_max = tmp;
      }
  }

  __shared__ float s_data[32];

  for (int64_t i = 1; i < 32; i *= 2) {
    float tmp = __shfl_xor_sync(unsigned(-1), local_max, i);
    if (tmp < local_max) {
        local_max = tmp;
    }
  }
  if (lane_id == 0) {
    s_data[warp_id] = local_max;
    s_data[warp_id] = local_max;
  }
  __syncthreads();
  float max = -INFINITY;
  if (lane_id < 8) {
    max = s_data[lane_id];
  }

  for (int64_t i = 1; i < 8; i *= 2) {
    float tmp =
        __shfl_xor_sync(unsigned(-1), max, i);
    if (tmp > max) {
        max = tmp;
    }
  }
  if (thread_id == 0) {
      output[output_dim] = max;
  }
}


int main() {
  const int d0 = 1;
  const int d1 = 256;
  const int d2 = 768;
  float *input, *mul, *param, *output;
  uint64_t mem_size = 1024 * sizeof(float);
  checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&input), mem_size));
  checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&output), mem_size));
  checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&mul), mem_size));
  checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&param), mem_size));
  float* h_init = reinterpret_cast<float*>(malloc(mem_size));
  for (int64_t i = 0; i < 1024; i++) {
    h_init[i] = 0.25;
  }
  checkCudaErrors(hipMemcpy(input, h_init, mem_size, hipMemcpyDefault));
  checkCudaErrors(hipMemcpy(param, h_init, mem_size, hipMemcpyDefault));
  checkCudaErrors(hipMemset(output, 0, mem_size));
  checkCudaErrors(hipMemset(mul, 0, mem_size));

  free(h_init);
//   float yita = 0.0001;
//   float gamma = 1.3;
//   float beta = 1.3;

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  // warmup
  checkCudaErrors(hipEventRecord(start));
  int times = 20;
  for (int i = 0; i < times; i++)
    softmax1st<<<1, 256>>>(input, mul, param, output);
  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));
  float msec = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msec, start, stop));
  printf("execution time = %f\n", msec);
  return 0;
}
